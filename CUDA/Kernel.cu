#include "hip/hip_runtime.h"
__global__ Kernel_Sample(size_t numberOfThreads)
{
	size_t threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid > numberOfInputPoints - 1) return;  
}

void Calling(size_t numberOfThreads)
{
    	int mingridsize;
    	int threadblocksize;
    	checkCudaErrors(hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, Kernel_Sample, 0, 0));
    	auto gridsize = (numberOfThreads - 1) / threadblocksize;
    
	Kernel_Sample << <gridsize, threadblocksize, 0, stream >> > ();
}
